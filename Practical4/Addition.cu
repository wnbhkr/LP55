#include "hip/hip_runtime.h"
#include <iostream>

// Kernel function to add two large vectors
__global__ void vectorAddition(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int size = 1000000; // Size of the vectors
    int *h_a, *h_b, *h_c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory for host vectors
    h_a = new int[size];
    h_b = new int[size];
    h_c = new int[size];

    // Initialize host vectors
    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = size - i;
    }

    // Allocate memory for device vectors
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Verify results
    for (int i = 0; i < 10; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
